#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount < 2) {
        printf("需要至少两个GPU进行P2P测试，当前只有 %d 个GPU\n", deviceCount);
        return 0;
    }

    printf("检测 %d 个GPU的P2P支持情况:\n\n", deviceCount);

    for (int i = 0; i < deviceCount; ++i) {
        for (int j = 0; j < deviceCount; ++j) {
            if (i == j) continue;

            int canAccessPeer = 0;
            hipDeviceCanAccessPeer(&canAccessPeer, i, j);
            printf("GPU %d -> GPU %d: %s\n", i, j,
                   canAccessPeer ? "支持" : "不支持");

            if (canAccessPeer) {
                hipSetDevice(i);
                hipError_t err = hipDeviceEnablePeerAccess(j, 0);
                if (err == hipErrorPeerAccessAlreadyEnabled) {
                    printf("  已启用访问\n");
                } else if (err != hipSuccess) {
                    printf("  启用失败: %s\n", hipGetErrorString(err));
                } else {
                    printf("  启用成功\n");
                }
            }
        }
    }

    // 简单的内存拷贝测试（可选）
    int size = 1024 * sizeof(int);
    int *d0, *d1;
    hipSetDevice(0);
    hipMalloc(&d0, size);
    hipSetDevice(1);
    hipMalloc(&d1, size);

    hipSetDevice(0);
    hipMemcpyPeer(d0, 0, d1, 1, size); // 从GPU1拷贝到GPU0
    printf("\n执行cudaMemcpyPeer完成\n");

    hipFree(d0);
    hipSetDevice(1);
    hipFree(d1);

    return 0;
}
